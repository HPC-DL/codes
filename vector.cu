
#include <cstdlib>
#include <iostream>
#include<hip/hip_runtime.h>
#include<chrono>

using namespace std::chrono;
using namespace std;

// VectorAdd parallel function
__global__ void vectorAdd(int *a, int *b, int *result, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n) {
        result[tid] = a[tid] + b[tid];
    }
}

int main() {
    int *a, *b, *c;
    int *a_dev, *b_dev, *c_dev;
    int n = 1 << 11; //equivalent to 2^15; bit shift 1 15 time to the left

    a = new int[n];
    b = new int[n];
    c = new int[n];
    int *d = new int[n];
    int size = n * sizeof(int);
    hipMalloc(&a_dev, size);
    hipMalloc(&b_dev, size);
    hipMalloc(&c_dev, size);

    // Array initialization..You can use Randon function to assign values
    for (int i = 0; i < n; i++) {
        a[i] = rand() % 1000;
        b[i] = rand() % 1000;
        d[i] = a[i] + b[i];  // calculating serial addition
    }
    cout << "Given array A is =>\n";
    for (int i = 0; i < n; i++) {
        cout << a[i] << ", ";
    }
    cout << "\n\n";

    cout << "Given array B is =>\n";
    for (int i = 0; i < n; i++) {
        cout << b[i] << ", ";
    }
    cout << "\n\n";

    hipEvent_t start, end;

    hipEventCreate(&start);
    hipEventCreate(&end);

    hipMemcpy(a_dev, a, size, hipMemcpyHostToDevice);
    hipMemcpy(b_dev, b, size, hipMemcpyHostToDevice);
    int threads = 1024;
    int blocks = (n + threads - 1) / threads;
    hipEventRecord(start);

    // Parallel addition program
    vectorAdd<<<blocks, threads>>>(a_dev, b_dev, c_dev, n);

    hipEventRecord(end);
    hipEventSynchronize(end);

    float time = 0.0;
    hipEventElapsedTime(&time, start, end);

    hipMemcpy(c, c_dev, size, hipMemcpyDeviceToHost);

    // vector addition using CPU
    auto s_start = high_resolution_clock::now();
    for (int i = 0; i < n; i++) {
        d[i] = a[i] + b[i];  
    }
    auto s_stop = high_resolution_clock::now();


    cout << "CPU sum is =>\n";
    for (int i = 0; i < n; i++) {
        cout << d[i] << ", ";
    }
    cout << "\n\n";

    cout << "GPU sum is =>\n";
    for (int i = 0; i < n; i++) {
        cout << c[i] << ", ";
    }
    cout << "\n\n";

    auto duration = duration_cast<microseconds>(s_stop - s_start);
    time = time * 1000;

    cout << "Time taken for sequential vector addition(in microseconds): " << duration.count() << endl;
    cout << "\nTime taken for parallel vector addition(in microseconds): " << time << endl;

    return 0;
}


