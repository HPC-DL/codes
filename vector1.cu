
#include <hip/hip_runtime.h>
#define N 100000000

__global__ void vector_add(float *out, float *a, float *b, int n) {

	for(int i = 0; i < n; i++){
		
		out[i] = a[i] + b[i];
		
		}

	}
	
int main(){

	float *a, *b, *out;
	float *d_a, *d_b, *d_out;
	
	//Allocate memory
	a = (float*)malloc(sizeof(float) * N);
	hipMalloc((void**)&d_a, sizeof(float) * N);
	
	b = (float*)malloc(sizeof(float) * N);
	hipMalloc((void**)&d_b, sizeof(float) * N);
	
	out = (float*)malloc(sizeof(float) * N);
	hipMalloc((void**)&d_out, sizeof(float) * N);
	
	//initialize array
	for(int i=0; i < N; i++){
		
		a[i] = 1.0f;
		b[i] = 2.0f;
		
		}
	
	hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(d_out, out, sizeof(float) * N, hipMemcpyHostToDevice);
		
	vector_add<<<1,1>>>(d_out, d_a, d_b, N);

	}
